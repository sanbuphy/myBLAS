
#include <hip/hip_runtime.h>
__global__ void sgemmKernel2(const int M, const int N, const int K,
                             float *__restrict__ A, float *__restrict__ B,
                             float *__restrict__ C) {
  const int BM = 128;
  const int BN = 128;
  const int BK = 8;
  const int TM = 8;
  const int TN = 8;
  const int threadRow = threadIdx.x / (BN / TN);
  const int threadCol = threadIdx.x % (BN / TN);

  __shared__ float sharedA[BM * BK];
  __shared__ float sharedB[BK * BN];

  float resultPerThread[TM * TN] = {0.0};
  float vectorOuterA[TM] = {0.0};
  float vectorOuterB[TN] = {0.0};

  A += blockIdx.y * BM * K;
  B += blockIdx.x * BN;
  C += blockIdx.y * BM * N + blockIdx.x * BM;

  const int sharedARow = threadIdx.x / (BK / 4);
  const int sharedACol = threadIdx.x % (BK / 4);
  const int sharedBRow = threadIdx.x / (BN / 4);
  const int sharedBCol = threadIdx.x % (BN / 4);

  for (int dotOrder = 0; dotOrder < K; dotOrder += BK) {
    reinterpret_cast<float4 *>(&sharedA[sharedARow * BK + sharedACol * 4])[0] =
        reinterpret_cast<float4 *>(&A[sharedARow * K + sharedACol * 4])[0];

    reinterpret_cast<float4 *>(&sharedB[sharedBRow * BN + sharedBCol * 4])[0] =
        reinterpret_cast<float4 *>(&B[sharedBRow * N + sharedBCol * 4])[0];
    __syncthreads();

    A += BK;
    B += BK * N;

#pragma unroll
    for (int innerOuterProdOrder = 0; innerOuterProdOrder < BK;
         innerOuterProdOrder++) {
#pragma unroll
      for (int i = 0; i < TM; i++)
        vectorOuterA[i] =
            sharedA[(threadRow * TM + i) * BK + innerOuterProdOrder];
#pragma unroll
      for (int i = 0; i < TN; i++)
        vectorOuterB[i] =
            sharedB[innerOuterProdOrder * BN + (threadCol * TN + i)];
#pragma unroll
      for (int resultRow = 0; resultRow < TM; resultRow++)
        for (int resultCol = 0; resultCol < TN; resultCol++)
          resultPerThread[resultRow * TN + resultCol] +=
              vectorOuterA[resultRow] * vectorOuterB[resultCol];
    }
    __syncthreads();
  }

#pragma unroll
  for (int resultRow = 0; resultRow < TM; resultRow += 1)
#pragma unroll
    for (int resultCol = 0; resultCol < TN; resultCol += 4) {
      float4 tmp;
      tmp.x = resultPerThread[resultRow * TN + resultCol + 0];
      tmp.y = resultPerThread[resultRow * TN + resultCol + 1];
      tmp.z = resultPerThread[resultRow * TN + resultCol + 2];
      tmp.w = resultPerThread[resultRow * TN + resultCol + 3];
      reinterpret_cast<float4 *>(&C[(threadRow * TM + resultRow) * N +
                                    (threadCol * TN + resultCol)])[0] = tmp;
    }
}